#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }   
}