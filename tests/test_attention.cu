#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "../src/cuda/attention.cuh"

class AttentionTest : public ::testing::Test {
protected:
    void SetUp() override {
        // Allocate test data
        batch_size = 2;
        seq_len = 4;
        head_dim = 8;
        size_t size = batch_size * seq_len * head_dim * sizeof(float);
        
        hipHostMalloc(&h_query, size);
        hipHostMalloc(&h_key, size);
        hipHostMalloc(&h_value, size);
        hipHostMalloc(&h_output, size);
        
        hipMalloc(&d_query, size);
        hipMalloc(&d_key, size);
        hipMalloc(&d_value, size);
        hipMalloc(&d_output, size);
        
        // Initialize test data
        for (size_t i = 0; i < batch_size * seq_len * head_dim; i++) {
            h_query[i] = 1.0f;
            h_key[i] = 2.0f;
            h_value[i] = 3.0f;
        }
        
        // Copy data to device
        hipMemcpy(d_query, h_query, size, hipMemcpyHostToDevice);
        hipMemcpy(d_key, h_key, size, hipMemcpyHostToDevice);
        hipMemcpy(d_value, h_value, size, hipMemcpyHostToDevice);
    }
    
    void TearDown() override {
        // Free memory
        hipHostFree(h_query);
        hipHostFree(h_key);
        hipHostFree(h_value);
        hipHostFree(h_output);
        
        hipFree(d_query);
        hipFree(d_key);
        hipFree(d_value);
        hipFree(d_output);
    }
    
    int batch_size, seq_len, head_dim;
    float *h_query, *h_key, *h_value, *h_output;
    float *d_query, *d_key, *d_value, *d_output;
};

TEST_F(AttentionTest, BasicFunctionality) {
    // Run the attention kernel
    compute_attention(d_query, d_key, d_value, d_output,
                     batch_size, seq_len, head_dim);
    
    // Copy result back to host
    size_t size = batch_size * seq_len * head_dim * sizeof(float);
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    
    // Verify output (basic test)
    for (size_t i = 0; i < batch_size * seq_len * head_dim; i++) {
        EXPECT_EQ(h_output[i], 0.0f);  // Currently just checking placeholder implementation
    }
}

int main(int argc, char **argv) {
    testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
} 